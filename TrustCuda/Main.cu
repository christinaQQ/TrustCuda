#include "hip/hip_runtime.h"
#include <array>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include "Peer.h"
#include "SimpleEigentrustGPU.h"

int main(void)
{
	const int m = 10000;
	const double error = 0.0001;
	const unsigned int num_transactions = 200000;

	if (m == 0)
		std::abort();

	std::vector<Peer> peers;
	for (int i = 0; i < m; i++)
	{
		peers.push_back(Peer());
	}

	SimpleEigentrustGPU eigentrust(peers, error);
	Peer::generateInteractions(eigentrust.getPeers(), num_transactions);

	thrust::host_vector<double> C(pow(m, 2));
	eigentrust.computeMatrix(C.begin(), C.end(), peers.begin(), peers.end());

	thrust::device_vector<double> d_C = C;
	thrust::device_vector<double> d_e(m, 1 / static_cast<double>(m));
	thrust::device_vector<double> d_y(m);
	eigentrust.computeEigentrust(thrust::raw_pointer_cast(&d_C[0]), thrust::raw_pointer_cast(&d_e[0]), thrust::raw_pointer_cast(&d_y[0]));

	thrust::host_vector<double> e = d_e;

	std::cout << "Peer ID:\t Trust Value" << std::endl;
	for (auto i = peers.begin(); i != peers.end(); i++)
	{
		std::cout << i->getId() << ":\t" << i->getTrustValue() << std::endl;
	}
	std::cin.get();

	return 0;
}