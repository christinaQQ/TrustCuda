#include "hip/hip_runtime.h"
#include "SimpleEigentrustGPU.h"
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <hipblas.h>

bool SimpleEigentrustGPU::hasConverged(double * trust_vec_next, double * trust_vec_orig)
{
	std::vector<Peer>::size_type m = getPeers().size();
	thrust::device_vector<double> d_v(m);
	square<double>        unary_op;
	thrust::plus<double> binary_op;
	double init = 0;

	thrust::device_ptr<double> trust_vec_next_ptr = thrust::device_pointer_cast(trust_vec_next);
	thrust::device_ptr<double> trust_vec_orig_ptr = thrust::device_pointer_cast(trust_vec_orig);
	thrust::transform(trust_vec_next_ptr, trust_vec_next_ptr + m, trust_vec_orig_ptr, d_v.begin(), thrust::minus<double>());
	double norm = std::sqrt(thrust::transform_reduce(d_v.begin(), d_v.end(), unary_op, init, binary_op));
	return norm < Eigentrust::getError() ? true : false;
}

void SimpleEigentrustGPU::computeEigentrust(double * C, double * e, double * y)
{
	std::vector<Peer>::size_type m = getPeers().size();
	double alpha = 1;
	double beta = 0;
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	do{
		hipblasDgemv(handle, HIPBLAS_OP_T, m, m, &alpha, C, m, e, 1, &beta, y, 1);
		double * tmp = e;
		e = y;
		y = tmp;
	} while (!hasConverged(e, y));
	hipblasDestroy(handle);
	setTrustValues(e);
}

void SimpleEigentrustGPU::setTrustValues(double * dev_trust_vector)
{
	thrust::device_ptr<double> dev_trust_vec_ptr = thrust::device_pointer_cast(dev_trust_vector);
	thrust::device_vector<double> d_trust_vector(dev_trust_vec_ptr, dev_trust_vec_ptr + getPeers().size());
	thrust::host_vector<double> host_trust_vector = d_trust_vector;
	for (int i = 0; i < getPeers().size(); i++)
	{
		getPeers().at(i).setTrustValue(*(host_trust_vector.begin() + i));
	}
}